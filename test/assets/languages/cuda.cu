/******************************************************************************
 * This is a file header, not counted as comment.                             *
 ******************************************************************************/

// I'm a comment line.

#include <hip/hip_runtime.h>
#include <stdio.h>

/**
 * Say Hello! to the world (from a CUDA kernel).
 */
__global__ void helloWorld() {
    printf("Hello World!\n");
}

int main() { // I'm a mixed code-comment line (counted as code).
    helloWorld<<< 1, 1 >>>();
    hipDeviceSynchronize();
    return 0;
}
